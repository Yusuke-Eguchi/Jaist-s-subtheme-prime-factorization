
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>

#define target 2*3*5*10000

__global__ void kernel(int *A)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j;
	int flag = 0;
	if(i < *A && i > 1){
		for(j=2;sqrtf(i)>=j;j++){
			if(i % j == 0){
				flag = 1;
			}
		}
		if(*A % i == 0 && flag == 0){
			printf("%d ", i);
		}
	}
}

int main(){
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
    int *d_target, A = target;
    hipMalloc((void**)&d_target,sizeof(int));
	hipMemcpy(d_target,&A,sizeof(int),hipMemcpyHostToDevice);
	dim3 block(1024);
	dim3 grid((A+1023)/1024);
	hipEventRecord(start);
	kernel<<<grid,block>>>(d_target);
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);
	hipEventDestroy(start);
	hipEventDestroy(stop);
	hipFree(d_target);
	printf("\n");
	printf("%10.10f\n", milliseconds);
return 0;
}