
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>

#define target 2*2*2*3*3*3*5*5*5

__global__ void kernel(int *A)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j;
	int flag = 0;
	if(i < *A && i > 1){
		for(j=2;sqrtf(i)>=j;j++){
			if(i % j == 0){
				flag = 1;
			}
		}
		if(*A % i == 0 && flag == 0){
			printf("%d ", i);
		}
	}
}

int main(){
    int *d_target, A = target;
    hipMalloc((void**)&d_target,sizeof(int));
	hipMemcpy(d_target,&A,sizeof(int),hipMemcpyHostToDevice);
	dim3 block(1024);
	dim3 grid((A+1023)/1024);
	kernel<<<grid,block>>>(d_target);
	hipFree(d_target);
	printf("\n");
return 0;
}