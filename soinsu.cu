#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#include <unistd.h>
#include <sys/time.h>
#include <sys/times.h>

clock_t times_clock()
{
    struct tms t;
    return times(&t);
}


#define target 2*3*5*1000000000000

__global__ void kernel(long long *A)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j;
	int flag = 0;
	if(i < *A && i > 1){
		for(j=2;sqrtf(i)>=j;j++){
			if(i % j == 0){
				flag = 1;
			}
		}
		if(*A % i == 0 && flag == 0){
			printf("%d ", i);
		}
	}
}

int main(){
	clock_t t1, t2;
    t1 = times_clock();
    int *d_target;
	long long A = target;
    hipMalloc((void**)&d_target,sizeof(int));
	hipMemcpy(d_target,&A,sizeof(long long),hipMemcpyHostToDevice);
	dim3 block(1024);
	dim3 grid((A+1023)/1024);
	kernel<<<grid,block>>>(d_target);
	hipFree(d_target);
	printf("\n");
    t2 = times_clock();
    printf("%10.100f\n", (double)(t2 - t1) / sysconf(_SC_CLK_TCK));
return 0;
}