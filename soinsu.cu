#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>

#define target 2*3*5*10000000
#define SIZE 100

double get_cputime(void)
{ 
 struct timespec t;
 clock_gettime(CLOCK_REALTIME,&t);
 //clock_gettime(CLOCK_THREAD_CPUTIME_ID,&t);
 return t.tv_sec + (double)t.tv_nsec*1e-9;
}
double get_realtime(void)
{
 struct timespec t;
 clock_gettime(CLOCK_REALTIME,&t);
 return t.tv_sec + (double)t.tv_nsec*1e-9;
}
double get_tick(void){ return (double)1e-9; }

__global__ void kernel(int *A)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j;
	int flag = 0;
	if(i < *A && i > 1){
		for(j=2;sqrtf(i)>=j;j++){
			if(i % j == 0){
				flag = 1;
			}
		}
		if(*A % i == 0 && flag == 0){
			printf("%lld ", i);
		}
	}
}

int main(){
	double t1, t2;
	t1 = get_realtime();
    int *d_target, A = target;
    hipMalloc((void**)&d_target,sizeof(int));
	hipMemcpy(d_target,&A,sizeof(int),hipMemcpyHostToDevice);
	dim3 block(1024);
	dim3 grid((A+1023)/1024);
	kernel<<<grid,block>>>(d_target);
	hipFree(d_target);
	printf("\n");
	t2 = get_realtime();
    printf("%10.100f\n", (double)(t2 - t1));
return 0;
}