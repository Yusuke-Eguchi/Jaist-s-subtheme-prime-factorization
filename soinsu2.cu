
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>

#define target 2*3*5

__device__ int GCD(int *a, int *b)
{
	int c;
	if(*a == 0){
		return *b;
	} else {
		c = *b % *a;
		return GCD(&c, a);
	}
}

__global__ void kernel(int *A, int *d_B, int *d_count)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;
	int k;
	int a = i - j, b, flag = 0;
	if(i >= __powf(*A,0.5) + 1 && j >= __powf(*A,0.5) + 1 && a > 1 && i < *A && j < *A){
		if(i^2 % *A == j^2 % *A){
			b = GCD(&a, A);
			for(k=2;b>k;k++){
				if(b % k == 0){
					flag = 1;
				}
			}
			if(flag == 0 && b != 1){
				d_B[*d_count] = b;
				*d_count++;
			}
		}
	}
}

int main(){
    int *d_target, A = target, count, *d_count;
	int *d_B;
	int B[target];
	int i, j;
	for(i=0;i<A;i++){
		B[i] = 0;
	}
    hipMalloc((void**)&d_target,sizeof(int));
	hipMalloc((void**)&d_B,sizeof(int)*A);
	hipMalloc((void**)&d_count,sizeof(int));
	hipMemcpy(d_target,&A,sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(d_B,&B,sizeof(int)*A,hipMemcpyHostToDevice);
	hipMemcpy(d_count,&count,sizeof(int),hipMemcpyHostToDevice);
	dim3 block(32,32);
	dim3 grid((A+31)/32,(A+31)/32);
	kernel<<<grid,block>>>(d_target,d_B,d_count);
	hipMemcpy(&B,d_B,sizeof(int)*A,hipMemcpyDeviceToHost);
	hipMemcpy(&count,d_count,sizeof(int),hipMemcpyDeviceToHost);
	hipFree(d_target);
	hipFree(d_B);	
	hipFree(d_count);
	printf("%d\n", count);
	for(i=0;i<count;i++){
		if(B[i] != 0){
			printf("%d ", B[i]);
		}
	}
	for(i=0;i<count;i++){
		for(j=i+1;j<count;j++){
			if(B[i] == B[j]){
				B[j] = 0;
			}
		}
	}
	for(i=0;i<count;i++){
		if(B[i] != 0){
			printf("%d ", B[i]);
		}
	}
	printf("\n");
return 0;
}