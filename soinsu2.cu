
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>

#define target 2*2*2*3*3*3*5*5*5

__device__ int GCD(int *a, int *b)
{
	int c;
	if(*a == 0){
		return *b;
	} else {
		c = *b % *a;
		return GCD(&c, a);
	}
}

__global__ void kernel(int *A, int *d_B)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;
	int k;
	int a = i - j, b, flag = 0;
	if(i >= __powf(*A,0.5) + 1 && j >= __powf(*A,0.5) + 1 && a > 1 && i < *A && j < *A){
		if(i^2 % *A == j^2 % *A){
			b = GCD(&a, A);
			for(k=2;b>k;k++){
				if(b % k == 0){
					flag = 1;
				}
			}
			if(flag == 0 && b != 1){
				d_B[i*j+j] = b;
			}
		}
	}
}

int main(){
    int *d_target, A = target;
	int *d_B;
	static int B[target*target];
	int i, j;
	for(i=0;i<A*A;i++){
		B[i] = 0;
	}
    hipMalloc((void**)&d_target,sizeof(int));
	hipMalloc((void**)&d_B,sizeof(int)*A*A);
	hipMemcpy(d_target,&A,sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(d_B,&B,sizeof(int)*A*A,hipMemcpyHostToDevice);
	dim3 block(32,32);
	dim3 grid((A+31)/32,(A+31)/32);
	kernel<<<grid,block>>>(d_target,d_B);
	hipMemcpy(&B,d_B,sizeof(int)*A*A,hipMemcpyDeviceToHost);
	hipFree(d_target);
	hipFree(d_B);	
	for(i=0;i<A*A;i++){
		for(j=i+1;j<A*A;j++){
			if(B[i] == B[j]){
				B[j] = 0;
			}
		}
	}
	for(i=0;i<A*A;i++){
		if(B[i] != 0){
			printf("%d ", B[i]);
		}
	}
	printf("\n");
return 0;
}