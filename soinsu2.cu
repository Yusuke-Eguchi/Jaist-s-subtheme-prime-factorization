#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>

#define target 2*2*2*3*3*3*5*5*5

__device__ void kernel2(int *A){
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int x;
	if(i <= __sqrt(*A)){
		x = __sqrt(*A) + i;
	}
}

__global__ void kernel(int *A)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	float w;
	if(i <= __sqrt(*A)){
		for(j = 2; j <= __sqrt(*A); j++) {
			kernel2(*A);
            printf("%d ",i);
        }
	}
}

int main(){
    int *d_target, A = target;
    hipMalloc((void**)&d_target,sizeof(int));
	hipMemcpy(d_target,&A,sizeof(int),hipMemcpyHostToDevice);
	dim3 block(1024);
	dim3 grid((A+1023)/1024);
	kernel<<<grid,block>>>(d_target);
	hipFree(d_target);
	printf("\n");
return 0;
}