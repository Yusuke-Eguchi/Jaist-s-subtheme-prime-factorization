#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>

#define target 2*3*5*10^50

__device__ int GCD(long int *a, long int *b)
{
	long int c;
	if(*a == 0){
		return *b;
	} else {
		c = *b % *a;
		return GCD(&c, a);
	}
}

__global__ void kernel(long int *A, int *d_B)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;
	int k;
	int flag = 0;
	long int a = i - j, b;
	if(i >= __powf(*A,0.5) + 1 && j >= __powf(*A,0.5) + 1 && a > 1 && i < *A && j < *A){
		if(i^2 % *A == j^2 % *A){
			b = GCD(&a, A);
			for(k=2;b>k;k++){
				if(b % k == 0){
					flag = 1;
				}
			}
			if(flag == 0 && b != 1){
				d_B[i*j+j] = b;
			}
		}
	}
}

int main(){
    long int *d_target, A = target;
	long int *d_B, B[target*target];
	int i, j;
	for(i=0;i<A*A;i++){
		B[i] = 0;
	}
    hipMalloc((void**)&d_target,sizeof(long int));
	hipMalloc((void**)&d_B,sizeof(long int)*A*A);
	hipMemcpy(d_target,&A,sizeof(long int),hipMemcpyHostToDevice);
	hipMemcpy(d_B,&B,sizeof(long int)*A*A,hipMemcpyHostToDevice);
	dim3 block(32,32);
	dim3 grid((A+31)/32,(A+31)/32);
	kernel<<<grid,block>>>(d_target,d_B);
	hipMemcpy(&B,d_B,sizeof(long int)*A*A,hipMemcpyDeviceToHost);
	hipFree(d_target);
	hipFree(d_B);	
	for(i=0;i<A*A;i++){
		for(j=i+1;j<A*A;j++){
			if(B[i] == B[j]){
				B[j] = 0;
			}
		}
	}
	for(i=0;i<A*A;i++){
		if(B[i] != 0){
			printf("%d ", B[i]);
		}
	}
	printf("\n");
return 0;
}