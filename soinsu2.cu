#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>

#define target 2*2*2*3*3*3*5*5*5

__device__ int GCD(int a, int b)
{
	int c;	
	if(a == 0){
		return b;
	} else if(a> b){
		return GCD(b, a);
	}else{
		c = b % a;
		return GCD(c, a);
	}
}

__global__ void kernel(int A)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;
	if(i >= 1 && j >= 1 && i > j){
		if((i+__sqrt(A))^2 % A == (j + __sqrt(A))^2 % A){
			printf("%d ", GCD(i - j, A));
		}
	}
}

int main(){
    int *d_target, A = target;
    hipMalloc((void**)&d_target,sizeof(int));
	hipMemcpy(d_target,&A,sizeof(int),hipMemcpyHostToDevice);
	dim3 block(32,32);
	dim3 grid((A+31)/32, (A+31)/32);
	kernel<<<grid,block>>>(d_target);
	hipFree(d_target);
	printf("\n");
return 0;
}