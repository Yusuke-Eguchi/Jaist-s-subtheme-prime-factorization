#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>

#define target 2*2*2*3*3*3*5*5*5

__device__ void GCD(int *a, int *b)
{
	if(*a = 0){
		return *b;
	}
	else{
		return GCD(b%a, a)
	}
}

__global__ void kernel(int *A)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;
	int a;
	if(i >= 1 && j >= 1 && i > j){
		if((i+__sqrt(*A))^2 % *A == (j + __sqrt(*A))^2 % *A &&){
			a = GCD(i + __sqrt(*A), j + __sqrt(*A));
		}
	}
retuen 0;
}

int main(){
    int *d_target, A = target, a, b, i, j;
    hipMalloc((void**)&d_target,sizeof(int));
	hipMemcpy(d_target,&A,sizeof(int),hipMemcpyHostToDevice);
	dim3 block(32,32);
	dim3 grid((A+31)/32, (A+31)/32);
	(i, j, a, b) = kernel<<<grid,block>>>(d_target);
	hipFree(d_target);
	printf("\n");
return 0;
}