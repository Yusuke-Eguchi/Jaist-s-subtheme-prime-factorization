
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>

#define target 2*2*2*3*3*3*5*5*5

__device__ int GCD(int *a, int *b)
{
	int c;
	if(*a == 0){
		return *b;
	} else {
		c = *b % *a;
		return GCD(&c, a);
	}
}

__global__ void kernel(int *A)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;
	int k;
	int a = i - j, b, flag = 0;
	if(i >= __powf(*A,0.5) && j >= __powf(*A,0.5) && a > 1 && i < *A && j < *A){
		if(i^2 % *A == j^2 % *A){
			b = GCD(&a, A);
			for(k=2;b>k;k++){
				if(b % k == 0){
					flag = 1;
				}
			}
			if(flag == 0 && b != 1){
				printf("%d ", b);
			}
		}
	}
}

int main(){
    int *d_target, A = target;
    hipMalloc((void**)&d_target,sizeof(int));
	hipMemcpy(d_target,&A,sizeof(int),hipMemcpyHostToDevice);
	dim3 block(32,32);
	dim3 grid((A+31)/32,(A+31)/32);
	kernel<<<grid,block>>>(d_target);
	hipFree(d_target);
	printf("\n");
return 0;
}