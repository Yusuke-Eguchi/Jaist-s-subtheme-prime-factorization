
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>

#define target 2*2*2*3*3*3*5*5*5

__global__ void kernel(int *A)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	float w;
	if(i <= *A){
		while(w != 0) {
                    printf("%d ",i);
        }
	}
}

int main(){
    int *d_target, A = target;
    hipMalloc((void**)&d_target,sizeof(int));
	hipMemcpy(d_target,&A,sizeof(int),hipMemcpyHostToDevice);
	dim3 block(1024);
	dim3 grid((A+1023)/1024);
	kernel<<<grid,block>>>(d_target);
	hipFree(d_target);
	printf("\n");
return 0;
}