
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>
#include <unistd.h>
#include <sys/time.h>
#include <sys/times.h>

clock_t times_clock()
{
    struct tms t;
    return times(&t);
}

#define target 2*3*5*10000
#define SIZE 100

__host__ int GCD(int a, long long b)
{
	int c;
	if(a == 0){
		return b;
	} else {
		c = b % a;
		return GCD(c, a);
	}
}

__global__ void kernel(int *A, int *d_B, int *d_count)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;
	int a = i - j;
	if(i >= __powf(*A,0.5) && j >= __powf(*A,0.5) && a > 1 && i < *A && j < *A){
		if(i^2 % *A == j^2 % *A){
			if(*d_count < SIZE){
				d_B[*d_count] = a;
				*d_count = *d_count + 1;
			}
		}
	}
}

int main(){
	clock_t t1, t2;
    t1 = times_clock();
    int count = 0, *d_count;
	int  *d_target, A = target;
	int *d_B;
	int B[SIZE];
	int i, j, k;
	for(i=0;i<SIZE;i++){
		B[i] = 0;
	}
    hipMalloc((void**)&d_target,sizeof(int));
	hipMalloc((void**)&d_B,sizeof(int)*SIZE);
	hipMalloc((void**)&d_count,sizeof(int));
	hipMemcpy(d_target,&A,sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(d_B,&B,sizeof(int)*SIZE,hipMemcpyHostToDevice);
	hipMemcpy(d_count,&count,sizeof(int),hipMemcpyHostToDevice);
	dim3 block(32,32);
	dim3 grid((A+31)/32,(A+31)/32);
	kernel<<<grid,block>>>(d_target,d_B,d_count);
	hipMemcpy(&B,d_B,sizeof(int)*SIZE,hipMemcpyDeviceToHost);
	hipMemcpy(&count,d_count,sizeof(int),hipMemcpyDeviceToHost);
	hipFree(d_target);
	hipFree(d_B);	
	hipFree(d_count);
	for(i=0;i<SIZE;i++){
		B[i] = GCD(B[i], A);
	}
	for(i=0;i<SIZE;i++){
		for(k=2;sqrtf(B[i])>=k;k++){
			if(B[i] % k == 0){
				B[i] = 0;
			}
		}
	}
	for(i=0;i<SIZE;i++){
		for(j=i+1;j<count;j++){
			if(B[i] == B[j]){
				B[j] = 0;
			}
		}
	}
	for(i=0;i<SIZE;i++){
		if(B[i] > 1){
			printf("%lld ", B[i]);
		}
	}
	printf("\n");
    t2 = times_clock();
    printf("%10.100f\n", (double)(t2 - t1) / sysconf(_SC_CLK_TCK));
return 0;
}