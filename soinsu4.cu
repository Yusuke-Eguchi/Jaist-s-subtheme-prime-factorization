
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>

#define target 2*3*5*10000
#define SIZE 100

__host__ int GCD(int a, int b)
{
	int c;
	if(a == 0){
		return b;
	} else {
		c = b % a;
		return GCD(c, a);
	}
}

__global__ void kernel(int *A, int *d_B, int *d_count)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;
	int a = i - j;
	if(i >= __powf(*A,0.5) + 1 && j >= __powf(*A,0.5) + 1 && a > 1 && i < *A && j < *A){
		if(i^2 % *A == j^2 % *A){
			if(*d_count < SIZE){
				d_B[*d_count] = a;
				*d_count = *d_count + 1;
			}
		}
	}
}

int main(){
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
    int *d_target, A = target, count = 0, *d_count;
	int *d_B;
	int B[SIZE];
	int i, j, k;
	for(i=0;i<SIZE;i++){
		B[i] = 0;
	}
    hipMalloc((void**)&d_target,sizeof(int));
	hipMalloc((void**)&d_B,sizeof(int)*SIZE);
	hipMalloc((void**)&d_count,sizeof(int));
	hipMemcpy(d_target,&A,sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(d_B,&B,sizeof(int)*SIZE,hipMemcpyHostToDevice);
	hipMemcpy(d_count,&count,sizeof(int),hipMemcpyHostToDevice);
	dim3 block(32,32);
	dim3 grid((A+31)/32,(A+31)/32);
	hipEventRecord(start);
	kernel<<<grid,block>>>(d_target,d_B,d_count);
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);
	hipEventDestroy(start);
	hipEventDestroy(stop);
	hipMemcpy(&B,d_B,sizeof(int)*SIZE,hipMemcpyDeviceToHost);
	hipMemcpy(&count,d_count,sizeof(int),hipMemcpyDeviceToHost);
	hipFree(d_target);
	hipFree(d_B);	
	hipFree(d_count);
	for(i=0;i<SIZE;i++){
		B[i] = GCD(B[i], A);
	}
	for(i=0;i<SIZE;i++){
		for(k=2;sqrtf(B[i])>=k;k++){
			if(B[i] % k == 0){
				B[i] = 0;
			}
		}
	}
	for(i=0;i<SIZE;i++){
		for(j=i+1;j<SIZE;j++){
			if(B[i] == B[j]){
				B[j] = 0;
			}
		}
	}
	for(i=0;i<SIZE;i++){
		if(B[i] != 0 && B[i] != 1){
			printf("%d ", B[i]);
		}
	}
	printf("\n");
	printf("%10.10f\n", milliseconds);
return 0;
}