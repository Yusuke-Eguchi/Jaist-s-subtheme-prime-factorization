#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#include <unistd.h>
#include <sys/time.h>
#include <sys/times.h>

clock_t times_clock()
{
    struct tms t;
    return times(&t);
}

#define target 2*3*5*10000
#define SIZE 1000

__host__ int GCD(int a, int b)
{
	int c;
	if(a == 0){
		return b;
	} else {
		c = b % a;
		return GCD(c, a);
	}
}

__global__ void kernel(long long *A, int *d_B, int *d_count)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;
	int a = i - j, b;
	if(i >= __powf(*A,0.5) + 1 && j >= __powf(*A,0.5) + 1 && a > 1 && i < *A && j < *A){
		if(i^2 % *A == j^2 % *A){
			if(*d_count > SIZE){
				b = a;
				b++;
			}
		}
	}
}

int main(){
	clock_t t1, t2;
    t1 = times_clock();
    int *d_target, count = 0, *d_count;
	long long  A = target;
	int *d_B;
	int B[SIZE];
	int i, j, k;
	for(i=0;i<SIZE;i++){
		B[i] = 0;
	}
    hipMalloc((void**)&d_target,sizeof(int));
	hipMalloc((void**)&d_B,sizeof(int)*SIZE);
	hipMalloc((void**)&d_count,sizeof(int));
	hipMemcpy(d_target,&A,sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(d_B,&B,sizeof(int)*SIZE,hipMemcpyHostToDevice);
	hipMemcpy(d_count,&count,sizeof(int),hipMemcpyHostToDevice);
	dim3 block(32,32);
	dim3 grid((A+31)/32,(A+31)/32);
	kernel<<<grid,block>>>(d_target,d_B,d_count);
	hipMemcpy(&B,d_B,sizeof(int)*SIZE,hipMemcpyDeviceToHost);
	hipMemcpy(&count,d_count,sizeof(int),hipMemcpyDeviceToHost);
	hipFree(d_target);
	hipFree(d_B);	
	hipFree(d_count);
	for(i=0;i<SIZE;i++){
		B[i] = GCD(B[i], A);
	}
	for(i=0;i<SIZE;i++){
		for(k=2;sqrtf(B[i])>=k;k++){
			if(B[i] % k == 0){
				B[i] = 0;
			}
		}
	}
	for(i=0;i<SIZE;i++){
		for(j=i+1;j<SIZE;j++){
			if(B[i] == B[j]){
				B[j] = 0;
			}
		}
	}
	for(i=0;i<SIZE;i++){
		if(B[i] > 1){
			printf("%d ", B[i]);
		}
	}
	printf("\n");
    t2 = times_clock();
    printf("%10.100f\n", (double)(t2 - t1) / sysconf(_SC_CLK_TCK));
return 0;
}