
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>
#include <unistd.h>
#include <sys/time.h>
#include <sys/times.h>

clock_t times_clock()
{
    struct tms t;
    return times(&t);
}

#define target 2*3*5*10000
#define SIZE 100

__host__ int GCD(int a, int b)
{
	int c;
	if(a == 0){
		return b;
	} else {
		c = b % a;
		return GCD(c, a);
	}
}

__global__ void kernel(int *A, int *d_B)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;
	int a = i - j;
	if(i >= __powf(*A,0.5) + 1 && j >= __powf(*A,0.5) + 1 && a > 1 && i < *A && j < *A){
		if(i^2 % *A == j^2 % *A){
			d_B[sizeof(d_B) / sizeof(int)] = a;
		}
	}
}

int main(){
	clock_t t1, t2;
	t1 = times_clock();
    int *d_target, A = target;
	int *d_B;
	int B[SIZE];
	int i, j, k;
    hipMalloc((void**)&d_target,sizeof(int));
	hipMalloc((void**)&d_B,sizeof(int)*SIZE);
	hipMemcpy(d_target,&A,sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(d_B,&B,sizeof(int)*SIZE,hipMemcpyHostToDevice);
	dim3 block(32,32);
	dim3 grid((A+31)/32,(A+31)/32);
	kernel<<<grid,block>>>(d_target,d_B);
	hipMemcpy(&B,d_B,sizeof(int)*SIZE,hipMemcpyDeviceToHost);
	hipFree(d_target);
	hipFree(d_B);	
	for(i=0;i<SIZE;i++){
		B[i] = GCD(B[i], A);
	}
	B[0] = -1;
	for(i=0;i<SIZE;i++){
		for(k=2;sqrtf(B[i])>=k;k++){
			if(B[i] % k == 0){
				B[i] = 0;
			}
		}
	}
	for(i=0;i<SIZE;i++){
		for(j=i+1;j<SIZE;j++){
			if(B[i] == B[j]){
				B[j] = 0;
			}
		}
	}
	for(i=0;i<SIZE;i++){
		if(B[i] > 1){
			printf("%d ", B[i]);
		}
	}
	printf("\n");
	t2 = times_clock();
    printf("%10.100f\n", (double)(t2 - t1) / sysconf(_SC_CLK_TCK));
return 0;
}