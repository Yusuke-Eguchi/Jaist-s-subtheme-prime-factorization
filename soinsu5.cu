#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#include <time.h>

struct timespec {
  time_t tv_sec; /* Seconds.  */
  long tv_nsec;  /* Nanoseconds.  */
};

#define target 2*3*5*10000
#define SIZE 100

__host__ int GCD(int a, int b)
{
	int c;
	if(a == 0){
		return b;
	} else {
		c = b % a;
		return GCD(c, a);
	}
}

__global__ void kernel(int *A, int *d_B)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;
	int a = i - j;
	if(i >= __powf(*A,0.5) + 1 && j >= __powf(*A,0.5) + 1 && a > 1 && i < *A && j < *A){
		if(i^2 % *A == j^2 % *A){
			d_B[sizeof(d_B) / sizeof(int)] = a;
		}
	}
}

int main(){
	struct timespec tp, start ,stop;
	clock_getres(CLOCK_REALTIME, struct timespec &tp);
	clock_gettime(CLOCK_REALTIME, struct timespec &start);
    int *d_target, A = target;
	int *d_B;
	int B[SIZE];
	int i, j, k;
    hipMalloc((void**)&d_target,sizeof(int));
	hipMalloc((void**)&d_B,sizeof(int)*SIZE);
	hipMemcpy(d_target,&A,sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(d_B,&B,sizeof(int)*SIZE,hipMemcpyHostToDevice);
	dim3 block(32,32);
	dim3 grid((A+31)/32,(A+31)/32);
	kernel<<<grid,block>>>(d_target,d_B);
	hipMemcpy(&B,d_B,sizeof(int)*SIZE,hipMemcpyDeviceToHost);
	hipFree(d_target);
	hipFree(d_B);	
	for(i=0;i<SIZE;i++){
		B[i] = GCD(B[i], A);
	}
	B[0] = -1;
	for(i=0;i<SIZE;i++){
		for(k=2;sqrtf(B[i])>=k;k++){
			if(B[i] % k == 0){
				B[i] = 0;
			}
		}
	}
	for(i=0;i<SIZE;i++){
		for(j=i+1;j<SIZE;j++){
			if(B[i] == B[j]){
				B[j] = 0;
			}
		}
	}
	for(i=0;i<SIZE;i++){
		if(B[i] > 1){
			printf("%d ", B[i]);
		}
	}
	printf("\n");
	clock_gettime(CLOCK_REALTIME, &stop);
    printf("%10.100f\n", (double)(stop - start));
return 0;
}