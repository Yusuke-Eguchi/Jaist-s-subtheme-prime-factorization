
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>

#define target 2*3*5*100000
#define SIZE 1000

__host__ int GCD(int a, int b)
{
	int c;
	if(a == 0){
		return b;
	} else {
		c = b % a;
		return GCD(c, a);
	}
}

__global__ void kernel(int *A, int *d_B)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;
	int a = i - j;
	if(i >= __powf(*A,0.5) + 1 && j >= __powf(*A,0.5) + 1 && a > 1 && i < *A && j < *A){
		if(i^2 % *A == j^2 % *A){
			if(sizeof(d_B) / sizeof(int) < SIZE ){
				d_B[sizeof(d_B) / sizeof(int)] = a;
			}
		}
	}
}

int main(){
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
    int *d_target, A = target;
	int *d_B;
	int B[SIZE];
	int i, j, k;
    hipMalloc((void**)&d_target,sizeof(int));
	hipMalloc((void**)&d_B,sizeof(int)*SIZE);
	hipMemcpy(d_target,&A,sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(d_B,&B,sizeof(int)*SIZE,hipMemcpyHostToDevice);
	dim3 block(32,32);
	dim3 grid((A+31)/32,(A+31)/32);
	hipEventRecord(start);
	kernel<<<grid,block>>>(d_target,d_B);
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);
	hipEventDestroy(start);
	hipEventDestroy(stop);
	hipMemcpy(&B,d_B,sizeof(int)*SIZE,hipMemcpyDeviceToHost);
	hipFree(d_target);
	hipFree(d_B);	
	for(i=0;i<SIZE;i++){
		B[i] = GCD(B[i], A);
	}
	for(i=0;i<SIZE;i++){
		for(k=2;sqrtf(B[i])>=k;k++){
			if(B[i] % k == 0){
				B[i] = 0;
			}
		}
	}
	for(i=0;i<SIZE;i++){
		for(j=i+1;j<SIZE;j++){
			if(B[i] == B[j]){
				B[j] = 0;
			}
		}
	}
	for(i=0;i<SIZE;i++){
		if(B[i] > 1){
			printf("%d ", B[i]);
		}
	}
	printf("\n");
	printf("%10.10f\n", milliseconds);
return 0;
}