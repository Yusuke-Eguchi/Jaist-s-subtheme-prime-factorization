#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#include <unistd.h>
#include <sys/time.h>
#include <sys/times.h>

clock_t times_clock()
{
    struct tms t;
    return times(&t);
}

#define target 2*3*5*1000000000000
#define SIZE 1000

__host__ int GCD(int a, int b)
{
	int c;
	if(a == 0){
		return b;
	} else {
		c = b % a;
		return GCD(c, a);
	}
}

__global__ void kernel(long long *A, int *d_B)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;
	int a = i - j;
	if(i >= __powf(*A,0.5) + 1 && j >= __powf(*A,0.5) + 1 && a > 1 && i < *A && j < *A){
		if(i^2 % *A == j^2 % *A){
			if(sizeof(d_B) / sizeof(long long) < SIZE ){
				d_B[sizeof(d_B) / sizeof(long long)] = a;
			}
		}
	}
}

int main(){
	clock_t t1, t2;
	t1 = times_clock();
    int *d_target；
	long long A = target;
	int *d_B;
	int B[SIZE];
	int i, j, k;
    hipMalloc((void**)&d_target,sizeof(long long));
	hipMalloc((void**)&d_B,sizeof(int)*SIZE);
	hipMemcpy(d_target,&A,sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(d_B,&B,sizeof(int)*SIZE,hipMemcpyHostToDevice);
	dim3 block(32,32);
	dim3 grid((A+31)/32,(A+31)/32);
	kernel<<<grid,block>>>(d_target,d_B);
	float milliseconds = 0;
	hipMemcpy(&B,d_B,sizeof(int)*SIZE,hipMemcpyDeviceToHost);
	hipFree(d_target);
	hipFree(d_B);	
	for(i=0;i<SIZE;i++){
		B[i] = GCD(B[i], A);
	}
	for(i=0;i<SIZE;i++){
		for(k=2;sqrtf(B[i])>=k;k++){
			if(B[i] % k == 0){
				B[i] = 0;
			}
		}
	}
	for(i=0;i<SIZE;i++){
		for(j=i+1;j<SIZE;j++){
			if(B[i] == B[j]){
				B[j] = 0;
			}
		}
	}
	for(i=0;i<SIZE;i++){
		if(B[i] > 1){
			printf("%d ", B[i]);
		}
	}
	printf("\n");
	t2 = times_clock();
    printf("%10.100f\n", (double)(t2 - t1) / sysconf(_SC_CLK_TCK));
return 0;
}